#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <iostream>

#define BLOCKS 1
#define THREADS 1024
#define ITER_TIMES 100

__device__ int vector[THREADS * BLOCKS];
__device__ int last_run;

__global__ void fill(){

    for (int i = 0; i < 1024; i++)
        vector[i] = i * 2;

}

__global__ void read_last_run(float time){

    //debugging measure
    if (ITER_TIMES == 1)
        printf("Last Run Result: %d and took: %f milliseconds\n", last_run, time);
    
    //reset value
    last_run = -1;

}

__global__ void parallel_redution_warp(){

    __shared__ int sum;
    sum = 0;

    //get our idex (assume 1d grids only)
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    int item = vector[idx];
    int warp_lane = idx - ((int)(threadIdx.x / 32) * 32);

    //use warp shuffle to add items
    for (int offset = 16; offset > 0; offset /= 2)
        item += __shfl_down_sync(0xffffffff, item, offset);

    //sync
    __syncthreads();
    
    //have each warp store their value
    if (warp_lane == 0){
        atomicAdd(&sum, item);
    }

    if (blockIdx.x == 0 && warp_lane == 0)
        last_run = sum;
    
    return;
}

__global__ void parallel_redution_block(){

    __shared__ volatile int shared_vector[THREADS];

    //get our idex (assume 1d grids only)
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);

    //sync
    __syncthreads();

    //load one element into shared memory as shown in slides
    shared_vector[idx] = vector[idx];

    //sync
    __syncthreads();

    //do reduction as shown in slides
    for (int i = 1; i < blockDim.x; i *= 2){

        //more volta-needed sync
        __syncthreads();

        if (!(idx % (2*i)))
            shared_vector[idx] += shared_vector[idx + i];

        //more volta-needed sync
        __syncthreads();
    }

    //write
    if (idx == 0)
        last_run = shared_vector[0];

    return;
}

__global__ void parallel_redution_block_branchless(){

    __shared__ volatile int shared_vector[THREADS];

    //get our idex (assume 1d grids only)
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);

    //sync
    __syncthreads();

    //load one element into shared memory as shown in slides
    shared_vector[idx] = vector[idx];

    //sync
    __syncthreads();

    //do reduction as shown in slides
    for (int i = 1; i < blockDim.x; i *= 2){
        int index = threadIdx.x * i * 2;

        //more volta-needed sync
        __syncthreads();

        if (index < blockDim.x)
            shared_vector[index] += shared_vector[index + i];

        //more volta-needed sync
        __syncthreads();
    }

    //write
    if (idx == 0)
        last_run = shared_vector[0];

    return;
}

__global__ void parallel_redution_block_interleaved(){
    
    __shared__ volatile int shared_vector[THREADS];

    //get our idex (assume 1d grids only)
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);

    //sync
    __syncthreads();

    //load one element into shared memory as shown in slides
    shared_vector[idx] = vector[idx];

    //sync
    __syncthreads();

    //do reduction as shown in slides
    for (int i = blockDim.x/2; i > 0; i >>= 1){

        //more volta-needed sync
        __syncthreads();

        if (idx < i)
            shared_vector[idx] += shared_vector[idx + i];

        //more volta-needed sync
        __syncthreads();
    }

    //write
    if (idx == 0)
        last_run = shared_vector[0];

    return;
}



int main(){

    //timing stuff
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    float times[4][ITER_TIMES] = {0.0};
    std::string parallel_mode[] = {"Warp Reduction", "Block Reduction", "Block Reduction Branchless", "Block Reduction Sequential Addresses"};

    // Warp Reduction
    /*******************************************************************/
    for (int i = 0; i < ITER_TIMES; i++){

        //fill vector with random values
        fill<<<1,1>>>();
        hipDeviceSynchronize();

        //now reduce with warp
        hipEventRecord(start);
        parallel_redution_warp<<<1, 1024>>>();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        read_last_run<<<1,1>>>(milliseconds);
        times[0][i] = milliseconds;
        hipDeviceSynchronize();
    }
    /*******************************************************************/

    // Block Reduction
    /*******************************************************************/
    for (int i = 0; i < ITER_TIMES; i++){

        //fill vector with random values
        fill<<<1,1>>>();
        hipDeviceSynchronize();

        //now reduce with warp
        hipEventRecord(start);
        parallel_redution_block<<<1, 1024>>>();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        read_last_run<<<1,1>>>(milliseconds);
        times[1][i] = milliseconds;
        hipDeviceSynchronize();
    }
    /*******************************************************************/

    // Block Reduction Branchless
    /*******************************************************************/    
    for (int i = 0; i < ITER_TIMES; i++){

        //fill vector with random values
        fill<<<1,1>>>();
        hipDeviceSynchronize();

        //now reduce with warp
        hipEventRecord(start);
        parallel_redution_block_branchless<<<1, 1024>>>();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        read_last_run<<<1,1>>>(milliseconds);
        times[2][i] = milliseconds;
        hipDeviceSynchronize();
    }
    /*******************************************************************/

    // Block Reduction Interleaved
    /*******************************************************************/    
    for (int i = 0; i < ITER_TIMES; i++){

        //fill vector with random values
        fill<<<1,1>>>();
        hipDeviceSynchronize();

        //now reduce with warp
        hipEventRecord(start);
        parallel_redution_block_interleaved<<<1, 1024>>>();
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        read_last_run<<<1,1>>>(milliseconds);
        times[3][i] = milliseconds;
        hipDeviceSynchronize();
    }
    /*******************************************************************/

    //read out results
    if (ITER_TIMES != 0){
        for (int i = 0; i < 4; i++){

            float total_times = 0.0;

            for (int j = 0; j < ITER_TIMES; j++){
                total_times += times[i][j];
            } 

            total_times /= ITER_TIMES;
            std::cout << "Average running time for " << parallel_mode[i] << " " << total_times << " ms\n";
        } 
    }
}
